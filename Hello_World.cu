#include <stdio.h>
#include <hip/hip_runtime.h>



// #include <stdio.h>

// // CUDA kernel function to print from the GPU
// __global__ void helloFromGPU()
// {
//     printf("Hello World from the GPU!\n");
// }

// int main()
// {
//     // Print from the CPU
//     printf("Hello World from the CPU!\n");

//     // Launch kernel on the GPU with one block and one thread
//     helloFromGPU<<<1, 1>>>();

//     // Wait for the GPU to finish before returning
//     cudaDeviceSynchronize();

//     return 0;
// }


#define N 16 // thread

// CUDA kernel
__global__ void helloFromGPU(int *array)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    printf("Hello World in thread %d in block %d. Array value: %d\n", threadIdx.x, blockIdx.x, array[idx]);
}

int main()
{

    int threadsPerBlock = 4;
    int numBlocks = 3;

    int h_array[N] = {0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15};

    int *d_array;

    // allocate memory
    hipMalloc((void **)&d_array, N * sizeof(int));

    // Copy data from host to device
    hipMemcpy(d_array, h_array, N * sizeof(int), hipMemcpyHostToDevice);

    helloFromGPU<<<numBlocks, threadsPerBlock>>>(d_array);

    hipDeviceSynchronize();

    hipFree(d_array);

    return 0;
}


