#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 16 // thread

// CUDA kernel
__global__ void helloFromGPU(int *array)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    printf("Hello World in thread %d in block %d. Array value: %d\n", threadIdx.x, blockIdx.x, array[idx]);
}

int main()
{

    int threadsPerBlock = 4;
    int numBlocks = 3;

    int h_array[N] = {0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15};

    int *d_array;

    // allocate memory
    hipMalloc((void **)&d_array, N * sizeof(int));

    // Copy data from host to device
    hipMemcpy(d_array, h_array, N * sizeof(int), hipMemcpyHostToDevice);

    helloFromGPU<<<numBlocks, threadsPerBlock>>>(d_array);

    hipDeviceSynchronize();

    hipFree(d_array);

    return 0;
}
