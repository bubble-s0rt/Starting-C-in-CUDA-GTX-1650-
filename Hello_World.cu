
#include <hip/hip_runtime.h>
#include <stdio.h>


__global__ void helloFromGPU()
{
    printf("Hello World from the GPU!\n");
}

int main()
{
   
    printf("Hello World from the CPU!\n");

    //launch kernel
    helloFromGPU<<<1, 1>>>();
    
    hipDeviceSynchronize();

    return 0;
}
